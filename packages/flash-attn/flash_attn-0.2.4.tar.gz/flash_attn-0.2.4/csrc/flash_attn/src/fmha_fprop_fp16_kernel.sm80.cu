/******************************************************************************
 * Copyright (c) 2011-2021, NVIDIA CORPORATION.  All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

#include "static_switch.h"
#include "fp16_switch.h"
#include "fmha.h"
#include "fmha_fprop_kernel_1xN.h"

void run_fmha_fp16_sm80(Launch_params<FMHA_fprop_params> &launch_params) {
    // FP16_SWITCH(launch_params.params.is_bf16, [&] {
    //     auto dprops = at::cuda::getCurrentDeviceProperties();
    //     if (launch_params.params.d <= 32) {
    //         if (launch_params.params.seqlen_k == 128) {
    //             using Kernel_traits = FMHA_kernel_traits<128, 32, 16, 1, 4, 0x08u, elem_type>;
    //             run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params);
    //         } else if (launch_params.params.seqlen_k >= 256) {
    //             using Kernel_traits = FMHA_kernel_traits<256, 32, 16, 1, 4, 0x08u, elem_type>;
    //             run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params);
    //         }
    //     } else if (launch_params.params.d <= 64) {
    //         if (launch_params.params.seqlen_k == 128) {
    //             using Kernel_traits = FMHA_kernel_traits<128, 64, 16, 1, 4, 0x08u, elem_type>;
    //             run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params);
    //         } else if (launch_params.params.seqlen_k >= 256) {
    //             using Kernel_traits = FMHA_kernel_traits<256, 64, 16, 1, 4, 0x08u, elem_type>;
    //             run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params);
    //         }
    //     } else if (launch_params.params.d <= 128) {
    //         // TD [2022-10-21]: Previously for SM80 we use block size 256 and keep K in shared memory
    //         // to reduce register spilling. However, that increases the smem usage from ~41KB to ~105KB,
    //         // reducing occupancy (only 1 kernel can be scheduled per SM instead of 2). This strategy gives
    //         // some speedup (6-10%) for large batch size, but slows things down for smal batch size.
    //         // Now that we have better parallelism (over seqlen_q), block size 128 is faster for small
    //         // batch size and only slightly slower (~3%) on large batch size.
    //         // For causal=True, block size 128 seems always faster (for small & large batch size).
    //         // So we're just gonna use block size 128 for simplicity.
    //         using Kernel_traits = FMHA_kernel_traits<128, 128, 16, 1, 4, 0x08u, elem_type>;
    //         run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params);
    //     }
    //     // if (launch_params.params.d == 64) {
    //     //     // using Kernel_traits = FMHA_kernel_traits<128, 64, 16, 1, 4, 0x08u, elem_type>;
    //     //     // using Kernel_traits = FMHA_kernel_traits<64, 64, 16, 1, 4, 0x08u, elem_type>;
    //     //     // using Kernel_traits = FMHA_kernel_traits<512, 64, 16, 1, 8, 0x08u, elem_type>;
    //     //     using Kernel_traits = FMHA_kernel_traits<128, 64, 16, 1, 4, 0x08u, elem_type>;
    //     //     run_fmha_fp16_sm80_loop_<Kernel_traits>(launch_params);
    //     // }
    // });
    if (launch_params.params.d <= 32) {
        run_fmha_fwd_hdim32(launch_params);
    } else if (launch_params.params.d <= 64) {
        run_fmha_fwd_hdim64(launch_params);
    } else if (launch_params.params.d <= 128) {
        run_fmha_fwd_hdim128(launch_params);
    }
}