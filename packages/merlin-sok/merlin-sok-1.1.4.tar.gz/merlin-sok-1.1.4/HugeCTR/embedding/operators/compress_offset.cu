#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "HugeCTR/embedding/common.hpp"
#include "HugeCTR/include/utils.hpp"
#include "compress_offset.hpp"
#include "generic_lookup.cuh"
namespace embedding {

__global__ void compress_offset_kernel(const uint32_t *offset, int num, int stride,
                                       uint32_t *compressed_offset) {
  int thread_cnt = blockDim.x * blockDim.y;

  for (int tid = threadIdx.x + threadIdx.y * blockDim.x; tid < num; tid += thread_cnt) {
    compressed_offset[tid] = offset[tid * stride];
  }
}

CompressOffset::CompressOffset(std::shared_ptr<CoreResourceManager> core, int num_compressed_offset)
    : core_(core), num_compressed_offset_(num_compressed_offset) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());

  auto buffer_ptr = GetBuffer(core);
  compressed_offset_ =
      buffer_ptr->reserve({num_compressed_offset}, DeviceType::GPU, TensorScalarType::UInt32);
  buffer_ptr->allocate();
}

void CompressOffset::compute(const Tensor &offset, int stride, Tensor *compressed_offset) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  auto stream = core_->get_local_gpu()->get_stream();

  dim3 block_size(32, 8);

  compress_offset_kernel<<<1, block_size, 0, stream>>>(
      offset.get<uint32_t>(), num_compressed_offset_, stride, compressed_offset_.get<uint32_t>());

  *compressed_offset = compressed_offset_;
}

AverageCombiner::AverageCombiner(std::shared_ptr<CoreResourceManager> core, int num_gpus,
                                 int num_local_embedding, const std::vector<int> &ev_size_list,
                                 int universal_batch_size)
    : core_(core), num_gpus_(num_gpus), num_local_embedding_(num_local_embedding) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());

  int num_ev_size_sum = std::accumulate(ev_size_list.begin(), ev_size_list.end(), 0);
  auto buffer_ptr = GetBuffer(core);
  // TODO: The float emb vec can be reduced to the scale of num of local embedding
  float_emb_vec_ = buffer_ptr->reserve({universal_batch_size / num_gpus, num_ev_size_sum},
                                       DeviceType::GPU, TensorScalarType::Float32);
  buffer_ptr->allocate();
}

void AverageCombiner::compute(const Tensor &bucket_range, const Tensor &top_grad,
                              const Tensor &d_local_embedding_list, const Tensor &d_combiner_list,
                              const Tensor &d_ev_size_offset, int batch_size, int max_ev_size) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  int gpu_id = core_->get_global_gpu_id();
  auto stream = core_->get_local_gpu()->get_stream();
  int batch_size_per_gpu = batch_size / num_gpus_;

  DISPATCH_INTEGRAL_FUNCTION(bucket_range.dtype().type(), offset_t, [&] {
    DISPATCH_FLOAT_AND_HALF_FUNCTION(top_grad.dtype().type(), emb_t, [&] {
      const offset_t *bucket_range_ptr = bucket_range.get<offset_t>();
      const int *local_embedding_ptr = d_local_embedding_list.get<int>();
      const int *d_ev_size_offset_ptr = d_ev_size_offset.get<int>();
      const emb_t *top_grad_ptr = top_grad.get<emb_t>();
      const char *combiner_ptr = d_combiner_list.get<char>();
      float *float_emb_vec_ptr = float_emb_vec_.get<float>();
      int gpu_id = core_->get_global_gpu_id();

      auto multi_to_one_desc = make_MultiToOne<emb_t, float>(
          batch_size_per_gpu * num_local_embedding_, [=] __device__(int i) { return i; },
          [=] __device__(int i) {
            int bid = i % batch_size_per_gpu;
            int lookup_id = local_embedding_ptr[i / batch_size_per_gpu];

            if (combiner_ptr[lookup_id] == static_cast<char>(Combiner::Average)) {
              int start = batch_size * lookup_id + gpu_id * batch_size_per_gpu + bid;
              return static_cast<int>(bucket_range_ptr[start + 1] - bucket_range_ptr[start]);
            } else {
              return 1;
            }
          },
          [=] __device__(int i) {
            int lookup_id = local_embedding_ptr[i / batch_size_per_gpu];
            return d_ev_size_offset_ptr[lookup_id + 1] - d_ev_size_offset_ptr[lookup_id];
          },
          [=] __device__(int i) {
            int bid = i % batch_size_per_gpu;
            int lookup_id = local_embedding_ptr[i / batch_size_per_gpu];

            int ev_offset = d_ev_size_offset_ptr[lookup_id] * batch_size_per_gpu;
            int ev_size = d_ev_size_offset_ptr[lookup_id + 1] - d_ev_size_offset_ptr[lookup_id];
            return top_grad_ptr + ev_offset + bid * ev_size;
          },
          [=] __device__(int i) {
            int bid = i % batch_size_per_gpu;
            int lookup_id = local_embedding_ptr[i / batch_size_per_gpu];

            int ev_offset = d_ev_size_offset_ptr[lookup_id] * batch_size_per_gpu;
            int ev_size = d_ev_size_offset_ptr[lookup_id + 1] - d_ev_size_offset_ptr[lookup_id];
            return float_emb_vec_ptr + ev_offset + bid * ev_size;
          });
      copy_multi_to_one(multi_to_one_desc, max_ev_size, stream);
    });
  });
}

}  // namespace embedding