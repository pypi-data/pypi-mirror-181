#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <hipcub/hipcub.hpp>

#include "HugeCTR/include/utils.hpp"
#include "all2all_embedding_collection.hpp"
namespace embedding {
namespace tf {

namespace {

template <typename offset_t>
__global__ void reorder_row_lengths_kernel(const offset_t *row_lengths, int num_row_lengths,
                                           offset_t *bucket_range, int batch_size_per_gpu,
                                           int num_gpu, int num_embedding) {
  for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < num_row_lengths;
       tid += blockDim.x * gridDim.x) {
    int gpu_id = tid / (batch_size_per_gpu * num_embedding);
    int embedding_id = (tid / batch_size_per_gpu) % num_embedding;
    int batch_id = tid % batch_size_per_gpu;

    int reorder_id =
        embedding_id * batch_size_per_gpu * num_gpu + gpu_id * batch_size_per_gpu + batch_id;
    bucket_range[1 + reorder_id] = row_lengths[tid];
  }
  if (threadIdx.x + blockIdx.x * blockDim.x == 0) {
    bucket_range[0] = 0;
  }
}

template <typename key_t, typename offset_t>
__global__ void reorder_key_kernel(const key_t *key, const offset_t *row_offsets,
                                   int num_row_lengths, const offset_t *bucket_range,
                                   key_t *reorder_key, int batch_size_per_gpu, int num_gpu,
                                   int num_embedding) {
  for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < num_row_lengths;
       tid += blockDim.x * gridDim.x) {
    int gpu_id = tid / (batch_size_per_gpu * num_embedding);
    int embedding_id = (tid / batch_size_per_gpu) % num_embedding;
    int batch_id = tid % batch_size_per_gpu;

    int reorder_id =
        embedding_id * batch_size_per_gpu * num_gpu + gpu_id * batch_size_per_gpu + batch_id;
    offset_t start = (tid == 0) ? 0 : row_offsets[tid];
    offset_t end = row_offsets[tid + 1];
    for (offset_t r = 0; r < (end - start); ++r) {
      reorder_key[bucket_range[reorder_id] + r] = key[start + r];
    }
  }
}

}  // namespace

namespace swizzle_key {

void sparse_forward_per_gpu(std::shared_ptr<CoreResourceManager> core,
                            const std::vector<Tensor> &keys, const std::vector<Tensor> &row_lengths,
                            Tensor &key_all_gather_send_buffer,
                            Tensor &row_lengths_all_gather_send_buffer) {
  size_t key_bytes_offset = 0;
  size_t row_lengths_bytes_offset = 0;
  key_all_gather_send_buffer.get();
  for (size_t i = 0; i < keys.size(); ++i) {
    HCTR_LIB_THROW(hipMemcpyAsync(
        reinterpret_cast<char *>(key_all_gather_send_buffer.get()) + key_bytes_offset,
        keys[i].get(), keys[i].nbytes(), hipMemcpyDeviceToDevice,
        core->get_local_gpu()->get_stream()));
    key_bytes_offset += keys[i].nbytes();

    HCTR_LIB_THROW(
        hipMemcpyAsync(reinterpret_cast<char *>(row_lengths_all_gather_send_buffer.get()) +
                            row_lengths_bytes_offset,
                        row_lengths[i].get(), row_lengths[i].nbytes(), hipMemcpyDeviceToDevice,
                        core->get_local_gpu()->get_stream()));
    row_lengths_bytes_offset += row_lengths[i].nbytes();
  }
}
}  // namespace swizzle_key

namespace model_forward {

std::vector<size_t> get_model_comm_buffer_size(const UniformModelParallelEmbeddingMeta &meta,
                                               int num_gpus, int batch_size) {
  size_t num_ev_elements = 0;
  int batch_size_per_gpu = batch_size / num_gpus;
  for (int lookup_id : meta.h_local_lookup_id_list_) {
    int ev_size = meta.h_ev_size_list_[lookup_id];
    num_ev_elements += ev_size * batch_size_per_gpu;
  }
  return std::vector<size_t>(num_gpus, num_ev_elements);
}

void sparse_forward_per_gpu(std::shared_ptr<CoreResourceManager> core,
                            const UniformModelParallelEmbeddingMeta &meta,
                            const Tensor &key_all_gather_recv_buffer,
                            const Tensor &row_lengths_all_gather_recv_buffer, ILookup *emb_storage,
                            std::vector<Tensor> &emb_vec_model_buffer, int64_t *num_model_key,
                            int64_t *num_model_offsets, Tensor *ret_model_key,
                            Tensor *ret_model_offset) {
  HugeCTR::CudaDeviceContext context(core->get_device_id());

  int num_gpus = core->get_global_gpu_count();
  hipStream_t stream = core->get_local_gpu()->get_stream();
  int batch_size = row_lengths_all_gather_recv_buffer.get_num_elements() / meta.num_lookup_;

  Tensor keys, bucket_range;
  size_t num_keys = static_cast<size_t>(key_all_gather_recv_buffer.get_num_elements());
  // the shape of key_all_gather is (num_gpus, num_embedding,
  // batch_size_per_gpu) the shape of key is (num_embedding, batch_size)
  auto reorder_from_all_gather_input = [&] {
    Tensor all_gather_row_offsets;

    auto buffer_ptr = GetBuffer(core);
    keys = buffer_ptr->reserve({key_all_gather_recv_buffer.get_num_elements()},
                               key_all_gather_recv_buffer.device(),
                               key_all_gather_recv_buffer.dtype());
    bucket_range = buffer_ptr->reserve({row_lengths_all_gather_recv_buffer.get_num_elements() + 1},
                                       row_lengths_all_gather_recv_buffer.device(),
                                       row_lengths_all_gather_recv_buffer.dtype());
    all_gather_row_offsets = buffer_ptr->reserve(
        {row_lengths_all_gather_recv_buffer.get_num_elements() + 1},
        row_lengths_all_gather_recv_buffer.device(), row_lengths_all_gather_recv_buffer.dtype());
    buffer_ptr->allocate();

    auto get_bucket_range = [&] {
      DISPATCH_INTEGRAL_FUNCTION(row_lengths_all_gather_recv_buffer.dtype().type(), offset_t, [&] {
        constexpr int block_size = 256;
        int grid_size =
            (row_lengths_all_gather_recv_buffer.get_num_elements() - 1) / block_size + 1;

        reorder_row_lengths_kernel<<<grid_size, block_size, 0, stream>>>(
            row_lengths_all_gather_recv_buffer.get<offset_t>(),
            row_lengths_all_gather_recv_buffer.get_num_elements(), bucket_range.get<offset_t>(),
            batch_size / num_gpus, num_gpus, meta.num_lookup_);

        size_t temp_bytes = 0;
        Tensor temp_scan_storage;
        hipcub::DeviceScan::InclusiveSum(nullptr, temp_bytes, (offset_t *)nullptr, (offset_t *)nullptr,
                                      bucket_range.get_num_elements());
        temp_scan_storage =
            buffer_ptr->reserve(temp_bytes, DeviceType::GPU, TensorScalarType::Void);
        buffer_ptr->allocate();

        hipcub::DeviceScan::InclusiveSum(temp_scan_storage.get(), temp_bytes,
                                      bucket_range.get<offset_t>(), bucket_range.get<offset_t>(),
                                      bucket_range.get_num_elements(), stream);

        // HCTR_LIB_THROW(hipStreamSynchronize(stream));

        // std::vector<offset_t> gpu_bucket_range;
        // bucket_range.to(&gpu_bucket_range);
        // std::cout << "gpu_bucket_range:\n";
        // for (auto i : gpu_bucket_range) {
        //   std::cout << i << " ";
        // }
        // std::cout << "\n";
      });
    };

    auto scan_row_lengths = [&] {
      DISPATCH_INTEGRAL_FUNCTION(row_lengths_all_gather_recv_buffer.dtype().type(), offset_t, [&] {
        size_t temp_bytes = 0;
        Tensor temp_scan_storage;
        hipcub::DeviceScan::InclusiveSum(nullptr, temp_bytes, (offset_t *)nullptr, (offset_t *)nullptr,
                                      row_lengths_all_gather_recv_buffer.get_num_elements() + 1);
        temp_scan_storage =
            buffer_ptr->reserve(temp_bytes, DeviceType::GPU, TensorScalarType::Void);
        buffer_ptr->allocate();

        hipcub::DeviceScan::InclusiveSum(
            temp_scan_storage.get(), temp_bytes, row_lengths_all_gather_recv_buffer.get<offset_t>(),
            all_gather_row_offsets.get<offset_t>() + 1,
            row_lengths_all_gather_recv_buffer.get_num_elements(), stream);
        // HCTR_LIB_THROW(hipStreamSynchronize(stream));

        // std::vector<offset_t> gpu_row_lengths;
        // row_lengths_all_gather_recv_buffer.to(&gpu_row_lengths);
        // std::cout << "gpu_row_lengths:\n";
        // for (auto i : gpu_row_lengths) {
        //   std::cout << i << " ";
        // }
        // std::cout << "\n";

        // std::vector<offset_t> gpu_row_offsets;
        // all_gather_row_offsets.to(&gpu_row_offsets);
        // std::cout << "gpu_row_offsets:\n";
        // for (auto i : gpu_row_offsets) {
        //   std::cout << i << " ";
        // }
        // std::cout << "\n";
      });
    };

    auto reorder_key = [&] {
      DISPATCH_INTEGRAL_FUNCTION(key_all_gather_recv_buffer.dtype().type(), key_t, [&] {
        DISPATCH_INTEGRAL_FUNCTION(
            row_lengths_all_gather_recv_buffer.dtype().type(), offset_t, [&] {
              constexpr int block_size = 256;
              int grid_size =
                  (row_lengths_all_gather_recv_buffer.get_num_elements() - 1) / block_size + 1;
              reorder_key_kernel<<<grid_size, block_size, 0, stream>>>(
                  key_all_gather_recv_buffer.get<key_t>(), all_gather_row_offsets.get<offset_t>(),
                  row_lengths_all_gather_recv_buffer.get_num_elements(),
                  bucket_range.get<offset_t>(), keys.get<key_t>(), batch_size / num_gpus, num_gpus,
                  meta.num_lookup_);
              // HCTR_LIB_THROW(hipStreamSynchronize(stream));

              // std::vector<key_t> gpu_all_gather_key;
              // key_all_gather_recv_buffer.to(&gpu_all_gather_key);
              // std::cout << "gpu_all_gather_key:\n";
              // for (auto i : gpu_all_gather_key) {
              //   std::cout << i << " ";
              // }
              // std::cout << "\n";

              // std::vector<offset_t> gpu_reorder_key;
              // keys.to(&gpu_reorder_key);
              // std::cout << "gpu_reorder_key:\n";
              // for (auto i : gpu_reorder_key) {
              //   std::cout << i << " ";
              // }
              // std::cout << "\n";
            });
      });
    };

    get_bucket_range();
    scan_row_lengths();
    reorder_key();
  };
  reorder_from_all_gather_input();

  DataType key_type = key_all_gather_recv_buffer.dtype();
  ModelIndexCalculation model_index_calculation_ =
      ModelIndexCalculation(core, meta.num_local_lookup_, meta.num_local_hotness_,
                            meta.hotness_sum_, batch_size, key_type);

  Tensor model_key, model_offsets;
  size_t num_model_key_;
  model_index_calculation_.compute(keys, bucket_range, num_keys, meta.d_local_lookup_id_list_,
                                   meta.d_local_shard_id_list_, meta.d_local_num_shards_list_,
                                   batch_size, &model_key, &model_offsets, &num_model_key_);

  CompressOffset compress_offset_ = CompressOffset(core, meta.num_local_lookup_ + 1);
  Tensor num_key_per_lookup_offset;
  compress_offset_.compute(model_offsets, batch_size, &num_key_per_lookup_offset);

  HCTR_LIB_THROW(hipStreamSynchronize(stream));
  TensorList embedding_vec = TensorList(core.get(), key_all_gather_recv_buffer.get_num_elements(),
                                        DeviceType::GPU, TensorScalarType::Float32);
  emb_storage->lookup(model_key, num_model_key_, num_key_per_lookup_offset,
                      meta.num_local_lookup_ + 1, meta.d_local_table_id_list_, embedding_vec);

  ModelForward model_forward_ = ModelForward(core, num_gpus, meta.h_local_lookup_id_list_);

  TensorList model_comm_buffer{core.get(), emb_vec_model_buffer, DeviceType::GPU,
                               emb_vec_model_buffer[0].dtype(), stream};
  model_forward_.compute(embedding_vec, model_offsets, model_comm_buffer,
                         meta.d_local_ev_size_list_, meta.d_local_ev_size_offset_, batch_size,
                         meta.max_ev_size_);

  *ret_model_key = model_key;
  *ret_model_offset = model_offsets;
  *num_model_key = static_cast<int64_t>(num_model_key_);
  *num_model_offsets = model_offsets.get_num_elements();
}

void copy_model_keys_and_offsets(std::shared_ptr<CoreResourceManager> core, const Tensor &model_key,
                                 const Tensor &model_offset, Tensor &tf_model_key,
                                 Tensor &tf_model_offsets) {
  HCTR_LIB_THROW(hipMemcpyAsync(tf_model_key.get(), model_key.get(), tf_model_key.nbytes(),
                                 hipMemcpyDeviceToDevice, core->get_local_gpu()->get_stream()));
  HCTR_LIB_THROW(hipMemcpyAsync(tf_model_offsets.get(), model_offset.get(),
                                 tf_model_offsets.nbytes(), hipMemcpyDeviceToDevice,
                                 core->get_local_gpu()->get_stream()));
}
}  // namespace model_forward

namespace network_forward {

void sparse_forward_per_gpu(std::shared_ptr<CoreResourceManager> core,
                            const UniformModelParallelEmbeddingMeta &meta,
                            const std::vector<Tensor> &emb_vec_network_buffer,
                            const std::vector<Tensor> &row_lengths,
                            std::vector<Tensor> &forward_emb_vec) {
  HugeCTR::CudaDeviceContext context(core->get_device_id());
  hipStream_t stream = core->get_local_gpu()->get_stream();
  int num_gpus = core->get_global_gpu_count();
  NetworkForward network_forward = NetworkForward(core, num_gpus);
  int batch_size_per_gpu = row_lengths[0].get_num_elements();
  int batch_size = batch_size_per_gpu * num_gpus;
  int global_gpu_id = core->get_global_gpu_id();

  TensorList row_lengths_buffer{core.get(), row_lengths, DeviceType::GPU, row_lengths[0].dtype(),
                                stream};
  TensorList network_comm_buffer{core.get(), emb_vec_network_buffer, DeviceType::GPU,
                                 emb_vec_network_buffer[0].dtype(), stream};
  TensorList output_buffer{core.get(), forward_emb_vec, DeviceType::GPU, forward_emb_vec[0].dtype(),
                           stream};
  network_forward.compute(row_lengths_buffer, meta.d_combiner_list_, network_comm_buffer,
                          meta.network_ids_, meta.network_gpu_ids_, meta.network_offsets_,
                          meta.network_dst_lookup_ids_, meta.network_ev_sizes_,
                          meta.network_ev_offsets_, output_buffer, meta.d_ev_size_offset_,
                          batch_size, meta.max_ev_size_);
}
}  // namespace network_forward

namespace network_backward {

void backward_per_gpu(std::shared_ptr<CoreResourceManager> core,
                      const UniformModelParallelEmbeddingMeta &meta,
                      const std::vector<Tensor> &top_grad, const std::vector<Tensor> &row_lengths,
                      std::vector<Tensor> &emb_vec_network_buffer) {
  HugeCTR::CudaDeviceContext context(core->get_device_id());
  hipStream_t stream = core->get_local_gpu()->get_stream();
  int num_gpus = core->get_global_gpu_count();
  int batch_size_per_gpu = row_lengths[0].get_num_elements();
  int batch_size = batch_size_per_gpu * num_gpus;

  NetworkBackward network_backward = NetworkBackward(core, num_gpus);

  TensorList row_lengths_buffer{core.get(), row_lengths, DeviceType::GPU, row_lengths[0].dtype(),
                                stream};
  TensorList network_comm_buffer{core.get(), emb_vec_network_buffer, DeviceType::GPU,
                                 emb_vec_network_buffer[0].dtype(), stream};
  TensorList top_grad_buffer{core.get(), top_grad, DeviceType::GPU, top_grad[0].dtype(), stream};

  network_backward.compute(row_lengths_buffer, meta.d_combiner_list_, top_grad_buffer,
                           meta.network_ids_, meta.network_gpu_ids_, meta.network_offsets_,
                           meta.network_dst_lookup_ids_, meta.network_ev_sizes_,
                           meta.network_ev_offsets_, network_comm_buffer, meta.d_ev_size_offset_,
                           batch_size, meta.max_ev_size_);
}
}  // namespace network_backward

namespace model_backward {

void sparse_backward_per_gpu(std::shared_ptr<CoreResourceManager> core,
                             const UniformModelParallelEmbeddingMeta &meta,
                             const std::vector<Tensor> &emb_vec_model_buffer,
                             const Tensor &model_key, const Tensor &model_offsets,
                             std::vector<int> *num_unique_key_per_table,
                             std::vector<int> *table_id_list, Tensor *ret_continous_unique_key,
                             Tensor *ret_continous_emb_vec) {
  HugeCTR::CudaDeviceContext context(core->get_device_id());
  int num_gpus = core->get_global_gpu_count();
  hipStream_t stream = core->get_local_gpu()->get_stream();
  int batch_size = (model_offsets.get_num_elements() - 1) / meta.num_local_lookup_;
  size_t num_model_key = static_cast<size_t>(model_key.get_num_elements());

  Tensor num_key_per_lookup_offset;
  CompressOffset compress_offset{core, meta.num_local_lookup_ + 1};
  compress_offset.compute(model_offsets, batch_size, &num_key_per_lookup_offset);

  ModelBackwardIndexCalculation model_backward_index_calculation_ = ModelBackwardIndexCalculation(
      core, num_gpus, meta.num_local_lookup_, meta.h_local_hotness_list_,
      meta.h_local_table_id_list_, meta.h_local_ev_size_list_, batch_size, model_key.dtype());

  Tensor continous_unique_key, wgrad_idx_offset, sorted_bucket_id_list, sorted_bucket_id_offset,
      d_table_id_list, num_unique_key_per_table_offset, continous_grad_emb_ev, coordinate_key,
      coordinate_wgrad_dst_idx;
  size_t num_unique_key;
  model_backward_index_calculation_.compute(
      model_key, num_model_key, model_offsets, num_key_per_lookup_offset,
      meta.d_local_table_id_list_, batch_size, &continous_unique_key, &num_unique_key,
      &wgrad_idx_offset, &sorted_bucket_id_list, &sorted_bucket_id_offset, &d_table_id_list,
      &num_unique_key_per_table_offset, &coordinate_key, &coordinate_wgrad_dst_idx);

  ModelBackward model_backward_ =
      ModelBackward(core, num_gpus, meta.num_local_lookup_, meta.h_local_hotness_list_,
                    meta.h_local_ev_size_list_, batch_size, meta.max_ev_size_, meta.num_sms_);

  TensorList model_comm_buffer{core.get(), emb_vec_model_buffer, DeviceType::GPU,
                               emb_vec_model_buffer[0].dtype(), stream};
  model_backward_.compute(model_comm_buffer, wgrad_idx_offset, sorted_bucket_id_list,
                          sorted_bucket_id_offset, num_unique_key, coordinate_key,
                          coordinate_wgrad_dst_idx, meta.d_local_ev_size_offset_, batch_size,
                          meta.max_ev_size_, num_model_key, &continous_grad_emb_ev);
  d_table_id_list.to(table_id_list, stream);
  *ret_continous_unique_key = continous_unique_key;
  *ret_continous_emb_vec = continous_grad_emb_ev;
  HCTR_LIB_THROW(hipStreamSynchronize(stream));
  std::vector<uint32_t> gpu_num_key_per_table_offset;
  num_unique_key_per_table_offset.to(&gpu_num_key_per_table_offset);

  num_unique_key_per_table->resize(d_table_id_list.get_num_elements());
  for (int i = 0; i < d_table_id_list.get_num_elements(); ++i) {
    (*num_unique_key_per_table)[i] =
        gpu_num_key_per_table_offset[i + 1] - gpu_num_key_per_table_offset[i];
  }
}

void copy_backward_key_and_emb_vec(std::shared_ptr<CoreResourceManager> core,
                                   const Tensor &continous_unique_key,
                                   const Tensor &continous_emb_vec, std::vector<Tensor> &unique_key,
                                   std::vector<Tensor> &emb_vec) {
  size_t nbytes_key_offsets = 0ul;
  size_t nbytes_emb_vec_offsets = 0ul;
  for (size_t i = 0; i < unique_key.size(); ++i) {
    HCTR_LIB_THROW(hipMemcpyAsync(
        unique_key[i].get(),
        reinterpret_cast<char *>(continous_unique_key.get()) + nbytes_key_offsets,
        unique_key[i].nbytes(), hipMemcpyDeviceToDevice, core->get_local_gpu()->get_stream()));
    HCTR_LIB_THROW(hipMemcpyAsync(
        emb_vec[i].get(),
        reinterpret_cast<char *>(continous_emb_vec.get()) + nbytes_emb_vec_offsets,
        emb_vec[i].nbytes(), hipMemcpyDeviceToDevice, core->get_local_gpu()->get_stream()));
    nbytes_key_offsets += unique_key[i].nbytes();
    nbytes_emb_vec_offsets += emb_vec[i].nbytes();
  }
}
}  // namespace model_backward
}  // namespace tf
}  // namespace embedding
