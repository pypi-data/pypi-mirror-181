#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <general_buffer2.hpp>
#include <optimizers/ftrl_optimizer.hpp>
#include <utils.cuh>
#include <utils.hpp>

namespace HugeCTR {

namespace {

template <typename T>
__global__ void ftrl_update_kernel(int len, float* weight, T* z, T* n, const T* wgrad, float alpha,
                                   float beta, float lambda1, float lambda2, float scaler) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    float gi = TypeConvertFunc<float, T>::convert(wgrad[i]) / scaler;
    float ni_new = TypeConvertFunc<float, T>::convert(n[i]) + gi * gi;
    float zi = TypeConvertFunc<float, T>::convert(z[i]) + gi +
               (sqrt(TypeConvertFunc<float, T>::convert(n[i])) - sqrt(ni_new)) *
                   TypeConvertFunc<float, T>::convert(weight[i]) / alpha;
    float x = lambda1 * (1.0f - 2.0f * signbit(zi)) - zi;
    float y = sqrt(ni_new) / alpha + lambda2;
    n[i] = TypeConvertFunc<T, float>::convert(ni_new);
    z[i] = TypeConvertFunc<T, float>::convert(zi);
    weight[i] = x / y * signbit(lambda1 - abs(zi));
  }
}

}  // namespace

template <typename T>
FtrlOptimizer<T>::FtrlOptimizer(const Tensor2<float>& weight_main, const Tensor2<T>& wgrad,
                                const std::shared_ptr<BufferBlock2<T>>& opt_buf,
                                const std::shared_ptr<GPUResource>& gpu_resource,
                                float learning_rate, float beta, float lambda1, float lambda2,
                                float scaler)
    : Optimizer(weight_main, gpu_resource, learning_rate, scaler),
      wgrad_(wgrad),
      beta_(beta),
      lambda1_(lambda1),
      lambda2_(lambda2) {
  if (weight_main_.get_num_elements() != wgrad_.get_num_elements()) {
    HCTR_OWN_THROW(Error_t::WrongInput, "weight->get_num_elements() != wgrad->get_num_elements()");
  }
  opt_buf->reserve({weight_main.get_num_elements()}, &n_);
  opt_buf->reserve({weight_main.get_num_elements()}, &z_);
}

template <typename T>
void FtrlOptimizer<T>::initialize() {
  HCTR_LIB_THROW(
      hipMemsetAsync(n_.get_ptr(), 0, n_.get_size_in_bytes(), gpu_resource_->get_stream()));
  HCTR_LIB_THROW(
      hipMemsetAsync(z_.get_ptr(), 0, z_.get_size_in_bytes(), gpu_resource_->get_stream()));
}

template <typename T>
void FtrlOptimizer<T>::update() {
  CudaDeviceContext context(get_device_id());

  const size_t len = weight_main_.get_num_elements();
  constexpr size_t block_dim = 256;
  const size_t grid_dim = (len - 1) / block_dim + 1;

  // const float alpha_t = lr_ * sqrt(1 - pow(beta2_, t_)) / (1 - pow(beta1_, t_));

  float* weight = weight_main_.get_ptr();

  T* z = z_.get_ptr();
  T* n = n_.get_ptr();
  const T* wgrad = wgrad_.get_ptr();
  ftrl_update_kernel<<<grid_dim, block_dim, 0, gpu_resource_->get_stream()>>>(
      len, weight, z, n, wgrad, lr_, beta_, lambda1_, lambda2_ + beta_ / lr_, scaler_);
#ifndef NDEBUG
  HCTR_LIB_THROW(hipDeviceSynchronize());
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template class FtrlOptimizer<float>;
template class FtrlOptimizer<__half>;

}  // namespace HugeCTR
